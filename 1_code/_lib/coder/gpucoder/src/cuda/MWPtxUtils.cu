// Copyright 2018-2019 The MathWorks, Inc.

#include "MWPtxUtils.hpp"

#ifdef MATLAB_MEX_FILE
#include "mex.h"
#endif

#include <sstream>

namespace mw_ptx_utils {

void initialize(const char* ptxData,
                const std::vector<const char*>& mangledNames,
                hipModule_t& module,
                std::vector<hipFunction_t>& functionHandles,
                bool reload) {
    if (reload || functionHandles.empty()) {
        loadPtx(ptxData, module);
        loadFunctionsFromModule(module, mangledNames, functionHandles);
    }
}

void loadPtx(const char* ptxData, hipModule_t& module) {    
    hipError_t loadResult = hipModuleLoadFatBinary(&module, ptxData);
    if (loadResult != hipSuccess){
        throwError("Failed to load CUDA module", loadResult);
    }
}

void loadFunctionsFromModule(const hipModule_t& module,
                             const std::vector<const char*>& mangledNames,
                             std::vector<hipFunction_t>& functionHandles) {
    for (std::vector<const char*>::const_iterator it = mangledNames.begin();
         it != mangledNames.end(); ++it) {
        hipFunction_t handle = loadFunction(module, *it);
        functionHandles.push_back(handle);
    }
}

hipFunction_t loadFunction(const hipModule_t& module, const char* mangledName) {
    hipFunction_t handle;
    hipError_t result = hipModuleGetFunction(&handle, module, mangledName);
    if (result != hipSuccess) {
        std::stringstream errorMessageStream;
        errorMessageStream <<
            "Unable to find function " <<
            mangledName <<
            " in the module. Load module failed with following error";
        std::string errorMessage(errorMessageStream.str());
        throwError(errorMessage, result);
        
    }
    return handle; 
}

hipError_t launchKernel(hipFunction_t kernel,
                      dim3 blocks,
                      dim3 threads,
                      void** args,
                      unsigned int sharedMem,
                      hipStream_t stream,
                      void** extra) {
    return hipModuleLaunchKernel(kernel, blocks.x, blocks.y, blocks.z,
                          threads.x, threads.y, threads.z,
                          sharedMem, stream, args, extra);
}

hipError_t launchKernelWithCheck(hipFunction_t kernel,
                               dim3 blocks,
                               dim3 threads,
                               void** args,
                               unsigned int sharedMem,
                               hipStream_t stream,
                               void** extra) {
    hipError_t result = launchKernel(kernel, blocks, threads, args, sharedMem, stream, extra);
    if (result != hipSuccess) {
        throwError("Error while evaluating kernel", result);
    }
    return result;
}

void throwError(const std::string& message) {
#ifdef MATLAB_MEX_FILE
    mexErrMsgIdAndTxt("gpucoder:ptxMexError", message.c_str());
#else
    throw std::runtime_error(message);
#endif
}

void throwError(const std::string& message, const hipError_t& error) {
    if (error == hipSuccess) {
        return;
    }
    const char* errorString;
    hipDrvGetErrorString(error, &errorString);

    std::string fullMessage = message + " - ";
    fullMessage += errorString;
    throwError(fullMessage);
}

void throwError(const std::string& message, const hipError_t& error) {
    if (error == hipSuccess) {
        return;
    }
    const char* errorString = hipGetErrorString(error);

    std::string fullMessage = message + " - ";
    fullMessage += errorString;
    throwError(fullMessage);
}

} // namespace mw_ptx_util
